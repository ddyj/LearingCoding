#include "hip/hip_runtime.h"
#include"cu_utils.h"

__device__ void distorted(float *dist_cu, const float x, const float y, 
                                                    float *dx, float *dy,int camera_id) {
    const float k1 = dist_cu[camera_id*5];
    const float k2 = dist_cu[camera_id*5+1];
    const float p1 = dist_cu[camera_id*5+2];
    const float p2 = dist_cu[camera_id*5+3];
    
    const float u2 = x * x;
    const float uv = x * y;
    const float v2 = y * y;
    const float r2 = u2 + v2;
    const float r4 = r2 * r2;
    const float r6 = r4 * r2;

    const float radial = (float(1) + k1 * r2 + k2 * r4);
    (*dx) = x * radial + float(2) * p1 * uv + p2 * (r2 + float(2) * u2) - x;
    (*dy) = y * radial + float(2) * p2 * uv + p1 * (r2 + float(2) * v2) - y;
    //printf("dx:%f - dy: %f\n",(*dx),(*dy));
}

__global__ void cudep2align(unsigned short* d_depth,
                                unsigned short* d_depth_final,
                                float* P1_cu, float *P2_cu,
                                float *R1_cu, float *dist_cu,
                                int width,int height,
                                int cur_id,int camera_num){
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;  //u
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;  //v

    if(x>=width || y>=height){return;}

    float dep = static_cast<float>(d_depth[cur_id*width*height+y*width+x])/1000.0;
    if(dep<=0){return ;}
    int camera_id = cur_id % camera_num;    
    
    float x1,y1,z1; //矫正相机坐标系    
    float x2,y2,z2;//原始相机系

    //真实相机坐标系->增加畸变
    float x3 = 0;
    float y3 = 0;

    float x4,y4;//加畸变后转到像素坐标系

    x1 = dep * (x-P1_cu[camera_id*12+2])/(P1_cu[camera_id*12]+1e-20);
    y1 = dep * (y-P1_cu[camera_id*12+6])/(P1_cu[camera_id*12+5]+1e-20);
    z1 = dep ;

    x2 = R1_cu[camera_id*9]*x1 + R1_cu[camera_id*9+1] *y1 + R1_cu[camera_id*9+2]*z1 ;
    y2 = R1_cu[camera_id*9+3]*x1 + R1_cu[camera_id*9+4] *y1 + R1_cu[camera_id*9+5]*z1;
    z2 = R1_cu[camera_id*9+6]*x1 + R1_cu[camera_id*9+7] *y1 + R1_cu[camera_id*9+8]*z1 ;
    float x_ = x2 / (z2 + 1e-20);
    float y_ = y2 / (z2 + 1e-20);
    distorted(dist_cu, x_, y_, &x3, &y3,camera_id);    
    x3 += x_;
    y3 += y_;

    x4 = P2_cu[camera_id*9] * x3 +  P2_cu[camera_id*9 + 2]; 
    y4 = P2_cu[camera_id*9+4] * y3 +  P2_cu[camera_id*9 + 5]; 

    if(x4>0 && x4<width && y4>0 && y4<height)
    {
        int u = x4;
        int v = y4;
        d_depth_final[cur_id*width*height+v*width+u] = static_cast<unsigned short>(z2*1000.0);
    }
}
                                

void dep2align(unsigned short* d_depth,
                                unsigned short* d_depth_final,
                                float* P1_cu,float *P2_cu,
                                float *R1_cu, float *dist_cu,
                                int m_width,int m_height,
                                int cur_id,int m_camera_num){

    //use 2*2的cuda核  通过xy二维控制图像
    dim3 threads(32, 32);
    unsigned int blockX = (threads.x + m_width - 1) / threads.x;
    unsigned int blockY = (threads.y + m_height - 1) / threads.y;
    dim3 blocks(blockX, blockY);

    cudep2align<<<blocks, threads>>>(d_depth,
                                            d_depth_final,
                                            P1_cu,P2_cu,R1_cu,dist_cu,
                                             m_width,m_height,
                                             cur_id,m_camera_num);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

}







__global__ void cudis2dep(float * dis_cu,unsigned short* dep_cu,float *T_cu,
                                                            int width,int height,
                                                            int cur_id,int camera_num){
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;  //u
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;  //v

    if(x>=width || y>=height){return;}

    float dis = static_cast<float>(dis_cu[cur_id*width*height+y*width+x]);
    int camera_id = cur_id % camera_num;
    //printf("%f - %f - %f \n",T_cu[camera_id*3],T_cu[camera_id*3+1],T_cu[camera_id*3+2]);
    float dep =  T_cu[camera_id*3] * T_cu[camera_id*3+1] / (dis + T_cu[camera_id*3+2]);
    //printf("%f - %f \n",dis,dep);
    if(dep<=5000 && dep>0){
        dep_cu[cur_id*width*height+y*width+x] = static_cast<unsigned short>(dep);
    }
}


void dis2dep(float * dis_cu,unsigned short* dep_cu,float *T_cu,
                            int width,int height,
                            int cur_id,int camera_num){
    //use 2*2的cuda核  通过xy二维控制图像
    dim3 threads(32, 32);
    unsigned int blockX = (threads.x + width - 1) / threads.x;
    unsigned int blockY = (threads.y + height - 1) / threads.y;
    dim3 blocks(blockX, blockY);

    cudis2dep<<<blocks, threads>>>(dis_cu,
                                             dep_cu,T_cu,
                                             width,height,
                                             cur_id,camera_num);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}



__global__ void kernelMyMemsetFloat(float *pdData,
                                    float value,
                                    unsigned int width,
                                    unsigned int height,
                                    unsigned int camera_num)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
    {
        return;
    }
    for (int  i = 0; i < camera_num; i++)
    {
        unsigned int offset = i*width*height +y * width + x;
        pdData[offset] = value;
    }
}
void cuMyMemsetFloat(float *pdData,
                                float value,
                                unsigned int width,
                                unsigned int height,
                                unsigned int camera_num)
{
    //use 2*2的cuda核  通过xy二维控制图像
    dim3 threads(32, 32);
    unsigned int blockX = (threads.x + width - 1) / threads.x;
    unsigned int blockY = (threads.y + height - 1) / threads.y;
    dim3 blocks(blockX, blockY);

    kernelMyMemsetFloat<<<blocks, threads>>>(pdData,
                                             value,
                                             width,
                                             height,
                                             camera_num);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
};

__global__ void kernelMyMemsetDouble(double *pdData,
                                    double value,
                                    unsigned int width,
                                    unsigned int height,
                                    unsigned int camera_num)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
    {
        return;
    }
    for (int  i = 0; i < camera_num; i++)
    {
        unsigned int offset = i*width*height +y * width + x;
        pdData[offset] = value;
    }
}
void cuMyMemsetDouble(double *pdData,
                                double value,
                                unsigned int width,
                                unsigned int height,
                                unsigned int camera_num)
{
    //use 2*2的cuda核  通过xy二维控制图像
    dim3 threads(32, 32);
    unsigned int blockX = (threads.x + width - 1) / threads.x;
    unsigned int blockY = (threads.y + height - 1) / threads.y;
    dim3 blocks(blockX, blockY);

    kernelMyMemsetDouble<<<blocks, threads>>>(pdData,
                                             value,
                                             width,
                                             height,
                                             camera_num);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
};

__global__ void kernelMyMemsetUC(unsigned char *pdData,
                                 unsigned char value,
                                 unsigned int width,
                                 unsigned int height,
                                 unsigned int camera_num)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
    {
        return;
    }
    for (int  i = 0; i < camera_num; i++)
    {
        unsigned int offset = i*width*height +y * width + x;
        pdData[offset] = value;
    }
}
void cuMyMemsetUC(unsigned char *pdData,
                             unsigned char value,
                             unsigned int width,
                             unsigned int height,
                             unsigned int camera_num)
{
    dim3 threads(32, 32);
    unsigned int blockX = (threads.x + width - 1) / threads.x;
    unsigned int blockY = (threads.y + height - 1) / threads.y;
    dim3 blocks(blockX, blockY);

    kernelMyMemsetUC<<<blocks, threads>>>(pdData,
                                          value,
                                          width,
                                          height,
                                          camera_num );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}
__global__ void kernelMyMemsetUS(unsigned short *pdData,
                                 unsigned short value,
                                 unsigned int width,
                                 unsigned int height,
                                 unsigned int camera_num)
{
    unsigned int u = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= width || v >= height)
    {
        return;
    }
    for (int  i = 0; i < camera_num; i++)
    {
        unsigned int offset = i*width*height +v * width + u;
        pdData[offset] = value;
    }
}
void cuMyMemsetUS(unsigned short *pdData,
                             unsigned short value,
                             unsigned int width,
                             unsigned int height,
                             unsigned int camera_num)
{
    
    dim3 threads(32, 32);
    
    unsigned int blockX = (threads.x + width - 1) / threads.x;
    unsigned int blockY = (threads.y + height - 1) / threads.y;
    dim3 blocks(blockX, blockY);
    kernelMyMemsetUS<<<blocks, threads>>>(pdData,
                                          value,
                                          width,
                                          height,
                                          camera_num );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}