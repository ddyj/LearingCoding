#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>


using namespace std;




struct Martix
{
    int width;
    int height;
    float* elements;
};

// 获取矩阵的(row, col)元素
__device__ float getElement(Martix* A,int row,int col){
    return A->elements[row*A->width + col];
}
// 获取矩阵的(row, col)元素
__device__ void setElement(Martix* A,int row,int col,float value){
    A->elements[row*A->width + col] = value;
}

//矩阵相乘
__global__ void matMulKernel(Martix* A,Martix* B,Martix* C){
    float Cvalue = 0.0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if(col >= A->width || row >= A->height|| col >= B->width || row >= B->height){
        return;
    }
    for(int i=0;i<A->width;++i){
        Cvalue += getElement(A,row,i) * getElement(B,i,col);
    }
    setElement(C,row,col,Cvalue); 
}

int main(){

    int width;
    int height;
    cin >> width;
    cin >> height;

    Martix* A;
    Martix* B;
    Martix* C;

    //申请托管内存
    hipMallocManaged((void**)&A,sizeof(Martix));
    hipMallocManaged((void**)&B,sizeof(Martix));
    hipMallocManaged((void**)&C,sizeof(Martix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void**)&A->elements,nBytes);
    hipMallocManaged((void**)&B->elements,nBytes);
    hipMallocManaged((void**)&C->elements,nBytes);

    //初始化数据
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;

    for(int i=0;i<width*height;++i){
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }
    //定义kernel的执行配置
    dim3 blockSize(32,32);
    dim3 gridSize((width + blockSize.x-1)/blockSize.x,(height + blockSize.y-1)/blockSize.y);

    matMulKernel<< <gridSize,blockSize >> >(A,B,C);
    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    //检查执行结果
    for(int i=0;i<width*height;++i){
        std::cout << C->elements[i]<< " ";
        if( (i+1) % width == 0){
            std::cout << "\b" << std::endl;
        }
    } 
    return 0;
}